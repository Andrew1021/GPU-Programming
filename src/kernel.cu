#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "kernel.cuh"

__constant__ int cudaVector[N];
__constant__ int cudaMatrix[N * N];
__constant__ int cudaMatrixMultiplied[N * N];

void cudaCaller(hipError_t command) 
{
    hipError_t const error = command;
    if (hipSuccess != error) {
        std::cout << hipGetErrorName(error) << "..." << hipGetErrorString(error) << std::endl;
    }
}

__global__ void multiplyKernel(int * _gpuReturn)
{
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex = threadIdx.x;
    if (threadIndex < N * N)
    {
        _gpuReturn[Index] = cudaMatrix[Index] * cudaVector[threadIndex % N];
    }
}

__global__ void addKernel(int * _gpuReturnResult)
{
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex = threadIdx.x;
    if (threadIndex < N)
    {
        for (int i = 0; i < N; i++)
        {
            _gpuReturnResult[Index] += cudaMatrixMultiplied[Index * N + i];
        }
    }
}

int multiply(const int* matrix, const int* vector, int * returnArray)
{
    hipDeviceProp_t properties;
    cudaCaller(hipGetDeviceProperties(&properties, 0));

    int* gpuReturnMatrix = NULL;
    int* gpuReturnResult = NULL;

    const int gpuVectorBufferSize = sizeof(int) * N;
    const int gpuMatrixBufferSize = sizeof(int) * N * N;

    cudaCaller(hipMalloc((void**)&gpuReturnMatrix, gpuMatrixBufferSize));
    cudaCaller(hipMalloc((void**)&gpuReturnResult, gpuVectorBufferSize));

    cudaCaller(hipMemcpyToSymbol(HIP_SYMBOL(cudaMatrix), matrix, gpuMatrixBufferSize));
    cudaCaller(hipMemcpyToSymbol(HIP_SYMBOL(cudaVector), vector, gpuVectorBufferSize));

    dim3 blockSettings((N * N + properties.maxThreadsPerBlock - 1) / properties.maxThreadsPerBlock);
    dim3 threadSettings(properties.maxThreadsPerBlock);
    multiplyKernel<<<blockSettings, threadSettings>>>(gpuReturnMatrix);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    cudaCaller(hipMemcpyToSymbol(HIP_SYMBOL(cudaMatrixMultiplied), gpuReturnMatrix, gpuMatrixBufferSize));

    dim3 blockSettings2((N + properties.maxThreadsPerBlock - 1) / properties.maxThreadsPerBlock);
    addKernel<<<blockSettings2, threadSettings>>>(gpuReturnResult);

    cudaCaller(hipMemcpy(returnArray, gpuReturnResult, gpuVectorBufferSize, hipMemcpyDeviceToHost));

    cudaCaller(hipFree(gpuReturnMatrix));

    return 0;
}