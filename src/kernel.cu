#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "kernel.cuh"

void cudaCaller(hipError_t command) 
{
    hipError_t const error = command;
    if (hipSuccess != error) {
        std::cout << hipGetErrorName(error) << "..." << hipGetErrorString(error) << std::endl;
    }
}

__global__ void multiplyKernel(int * _gpuReturnMatrix, int* _cudaVector, int* _cudaMatrix)
{
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex = threadIdx.x;
    if (threadIndex < N * N)
    {
        _gpuReturnMatrix[Index] = _cudaMatrix[Index] * _cudaVector[threadIndex % N];
    }
}

__global__ void addKernel(int* gpuReturnMatrix, int * _gpuReturnResult)
{
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex = threadIdx.x;
    if (threadIndex < N)
    {
        for (int i = 0; i < N; i++)
        {
            _gpuReturnResult[Index] += gpuReturnMatrix[Index * N + i];
        }
    }
}

int multiply(const int* matrix, const int* vector, int * returnArray)
{
    hipDeviceProp_t properties;
    properties.major = 5;
    int Device;
    cudaCaller(hipChooseDevice(&Device, &properties));
    cudaCaller(hipSetDevice(Device));
    cudaCaller(hipGetDeviceProperties(&properties, Device));

    int* returnArrayMultiplied = new int[N * N];

    int* gpuReturnMatrix = NULL;
    int* gpuReturnResult = NULL;
    int* gpuArrayMultiplied = NULL;

    int* cudaVector = NULL;
    int* cudaMatrix = NULL;

    const int gpuVectorBufferSize = sizeof(int) * N;
    const int gpuMatrixBufferSize = sizeof(int) * N * N;

    cudaCaller(hipMalloc((void**)&gpuReturnResult, gpuVectorBufferSize));

    cudaCaller(hipMalloc((void**)&gpuReturnMatrix, gpuMatrixBufferSize));
    cudaCaller(hipMalloc((void**)&cudaVector, gpuVectorBufferSize));
    cudaCaller(hipMalloc((void**)&cudaMatrix, gpuMatrixBufferSize));

    cudaCaller(hipMemcpy(cudaVector, vector, gpuVectorBufferSize, hipMemcpyHostToDevice));
    cudaCaller(hipMemcpy(cudaMatrix, matrix, gpuMatrixBufferSize, hipMemcpyHostToDevice));
    
    dim3 blockSettings((N * N + properties.maxThreadsPerBlock - 1) / properties.maxThreadsPerBlock);
    dim3 threadSettings(properties.maxThreadsPerBlock);
    multiplyKernel<<<blockSettings, threadSettings>>>(gpuReturnMatrix, cudaVector, cudaMatrix);

    cudaCaller(hipDeviceSynchronize());

    cudaCaller(hipMemcpy(returnArrayMultiplied, gpuReturnMatrix, gpuMatrixBufferSize, hipMemcpyDeviceToHost));

    cudaCaller(hipMalloc((void**)&gpuArrayMultiplied, gpuMatrixBufferSize));
    cudaCaller(hipMemcpy(gpuArrayMultiplied, returnArrayMultiplied, gpuMatrixBufferSize, hipMemcpyHostToDevice));

    dim3 blockSettings2((N + properties.maxThreadsPerBlock - 1) / properties.maxThreadsPerBlock);
    addKernel<<<blockSettings2, threadSettings>>>(gpuArrayMultiplied, gpuReturnResult);

    cudaCaller(hipMemcpy(returnArray, gpuReturnResult, gpuVectorBufferSize, hipMemcpyDeviceToHost));

    cudaCaller(hipFree(gpuReturnResult));
    cudaCaller(hipFree(gpuReturnMatrix));
    cudaCaller(hipFree(cudaVector));
    cudaCaller(hipFree(cudaMatrix));
    cudaCaller(hipFree(gpuArrayMultiplied));

    return 0;
}